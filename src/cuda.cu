#include "hip/hip_runtime.h"
#include "moderngpu/transform.hxx"
#include "moderngpu/memory.hxx"
#include "moderngpu/context.hxx"
#include <vector>
#include <iostream>
#include <chrono>
#include <algorithm>
using namespace std;

int find_geq_pow_2(int n) {
  int res = 1;
  while (res < n) res <<= 1;
  return res;
}

void gpusort(int n, long long* dev_a) {
  mgpu::standard_context_t context(false, 0);
  if (n <= 1) return;
  int m = find_geq_pow_2(n);
  mgpu::mem_t<long long> v_mem = mgpu::fill(std::numeric_limits<long long>::max(), m, context);
  long long *v = v_mem.data();
  mgpu::dtod(v, dev_a, n);

  for (int k = 2; k <= m; k <<= 1) { // loop subtask size
    for (int half = k >> 1; half > 0; half >>= 1) { // merging
      mgpu::transform(
        []MGPU_DEVICE(int index, int k, int half, long long *ar) {
          int j = index + half;
          if ((index & half) == 0) {
            if ((index & k) == 0) { //ascending
              if (ar[index] > ar[j]) {
                long long tmp = ar[index];
                ar[index] = ar[j];
                ar[j] = tmp;
              }
            } else {
              if (ar[index] < ar[j]) {
                long long tmp = ar[index];
                ar[index] = ar[j];
                ar[j] = tmp;
              }
            }
          }
        },
        m,
        context,
        k,
        half,
        v
      );
    }
  }

  mgpu::dtod(dev_a, v, n);
}


int main() {
  int n; std::cin >> n;
  std::vector<long long> v(n);
  for (auto &e : v) {
    std::cin >> e;
  }
  long long* dev_a;
  hipMalloc(&dev_a, n * sizeof(long long));
  hipMemcpy(dev_a, v.data(), n * sizeof(long long), hipMemcpyHostToDevice);
  
  auto start_time = std::chrono::high_resolution_clock::now();
  gpusort(n, dev_a);
  auto end_time = std::chrono::high_resolution_clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time);
  std::cout << "CUDA sort time: " << duration.count() << " milliseconds" << std::endl;
  
  std::vector<long long> sorted(n);
  hipMemcpy(sorted.data(), dev_a, n * sizeof(long long), hipMemcpyDeviceToHost);
  std::sort(v.begin(), v.end());
  bool ok = true;
  for (size_t i = 0; i < n; ++i) {
    if (v[i] != sorted[i]) ok = false;
  }
  if (ok) {
    std::cout << "OK - bitonic sort on " << n << " elements\n";
  } else {
    std::cout << "Error - bitonic sort on " << n << " elements\n";
  }
}